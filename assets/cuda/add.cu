#include <hip/hip_runtime.h>

#include <stdio.h>

#define BLOCK_NUM 8    //块数量
#define THREAD_NUM 64  // 每个块中的线程数
#define R_SIZE BLOCK_NUM *THREAD_NUM
#define M_SIZE R_SIZE *R_SIZE

__global__ void mat_add(int *mat1, int *mat2, int *result) {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  // 每个线程计算一行
  const int row = bid * THREAD_NUM + tid;
  for (int i = 0; i < R_SIZE; i++) {
    int index = row * R_SIZE + i;
    result[index] = mat1[index] * mat2[index];
  }
}

int main(int argc, char *argv[]) {
  int *mat1, *mat2, *result;
  int *g_mat1, *g_mat2, *g_mat_result;

  // 用一位数组表示二维矩阵
  mat1 = (int *)malloc(M_SIZE * sizeof(int));
  mat2 = (int *)malloc(M_SIZE * sizeof(int));
  result = (int *)malloc(M_SIZE * sizeof(int));

  // initialize
  for (int i = 0; i < M_SIZE; i++) {
    mat1[i] = rand() / 1000000;
    mat2[i] = rand() / 1000000;
    result[i] = 0;
  }

  hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
  hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
  hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

  hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

  mat_add<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

  hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE,
             hipMemcpyDeviceToHost);
  printf("mat1[0][0] is %d\tmat2[0][0] is %d\treault[0][0] is %d\n", mat1[0],
         mat2[0], result[0]);
  free(mat1);
  free(mat2);
  free(result);
}