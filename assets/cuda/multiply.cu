#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>

#define BLOCK_NUM 8    //块数量
#define THREAD_NUM 64  // 每个块中的线程数
#define R_SIZE BLOCK_NUM *THREAD_NUM
#define M_SIZE R_SIZE *R_SIZE

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  // 每个线程计算一行
  const int row = bid * THREAD_NUM + tid;
  for (int c = 0; c < R_SIZE; c++) {
    for (int n = 0; n < R_SIZE; n++) {
      result[row * R_SIZE + c] += mat1[row * R_SIZE + n] * mat2[n * R_SIZE + c];
    }
  }
}

int main(int argc, char *argv[]) {
  int *mat1, *mat2, *result;
  int *g_mat1, *g_mat2, *g_mat_result;

  // 用一位数组表示二维矩阵
  mat1 = (int *)malloc(M_SIZE * sizeof(int));
  mat2 = (int *)malloc(M_SIZE * sizeof(int));
  result = (int *)malloc(M_SIZE * sizeof(int));

  // initialize origin mat1
  for (int i = 0; i < M_SIZE; i++) {
    mat1[i] = rand() / 1000000;
    result[i] = 0;
  }
  for (int i = 0; i < R_SIZE; i++) {
    mat2[i * (R_SIZE + 1)] = 2;
  }

  hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
  hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
  hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

  hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

  mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

  hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE,
             hipMemcpyDeviceToHost);
  printf("mat1[0][0] is %d\tmat2[0][0] is %d\treault[0][0] is %d\n", mat1[0],
         mat2[0], result[0]);
  free(mat1);
  free(mat2);
  free(result);
}